
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 50

//  multi-kernel for being able to handle an array larger than 2048
// gpu can't return from the kernel
__global__ void Reduction(int* input, int operation){

// Sum Operation
	if (operation == 0) {
		for (unsigned int stride = 1;  stride <= blockDim.x; stride *= 2) {
		 	
		 __syncthreads();

			
		 	if (threadIdx.x % stride == 0) {
		 		input[2*threadIdx.x] += input[2*threadIdx.x+stride];
		 	}
		 		
		}	

	if (threadIdx.x == 0) {
		printf("Sum: %d\n", input[0]);
	}
	
	}

}

int main(){
	// allocate memory
	// int *input, *output; 
	int *input;
	
	hipMallocManaged(&input, sizeof(int) * SIZE);
	// cudaMallocManaged(&output, sizeof(int) * SIZE);

  	// initialize inputs
  	for (int i = 0; i < SIZE; i++) {
  		input[i] = 1;
   	}
	
	// Reduction(input, output, 0);
	// printf("Output: %d", Reduction(input, output, 0));

	Reduction<<<1, SIZE>>>(input, 0);

/*
	if SIZE < 2048{
		Reduction<<<1, SIZE>>>(input, output, 0);
	}
	
	else{
		Reduction<<<1, SIZE>>>(input, output, 0);
		// ReductionLarge<<<>>>();
	}
*/

	/*
	  // check results
 	 for (int i = 0; i < SIZE; i++) {
    	printf("%d ", output[i]);
  	 }
 	 printf("\n");
 	*/ 
 	printf("%s\n", hipGetErrorString(hipGetLastError()));

	hipFree(input);
	// cudaFree(output);

	return 0;
	
}
