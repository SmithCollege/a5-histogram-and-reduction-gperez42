
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 4096 // array size
#define BLOCKSIZE 1024 // number of threads per block

// Sources: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf, Microsoft Copilot for help managing memory issues

__global__ void Reduction(int* input, int operation) {
	 __shared__ int sdata[SIZE];
	// __shared__ int sdata[BLOCKSIZE * 2];
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	
	// Initialize shared memory with values from global memory
    sdata[threadIdx.x] = (i < SIZE) ? input[i] : 0;
    if ((i + BLOCKSIZE) < SIZE) {
        sdata[threadIdx.x + BLOCKSIZE] = input[i + BLOCKSIZE];
    } else {
        sdata[threadIdx.x + BLOCKSIZE] = 0;
    }
  	__syncthreads();
  	
    // Sum Operation
    if (operation == 0) {
        for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
            __syncthreads();
            if (threadIdx.x % stride == 0) {
                sdata[2 * threadIdx.x] += sdata[2 * threadIdx.x + stride];
            }
        }
    }
    
	if (operation == 1){
		for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
	        __syncthreads();
	        if (threadIdx.x % stride == 0) {
	            sdata[2 * threadIdx.x] *= sdata[2 * threadIdx.x + stride];
	        }
	    }
	}

	if (operation == 2) {
	    //max = sdata[0];
		for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
	    	__syncthreads();
		    //if (max < sdata[2 * threadIdx.x + stride]) {
		   if (sdata[2 * threadIdx.x] < sdata[2 * threadIdx.x + stride]) {
		        sdata[2 * threadIdx.x] = sdata[2 * threadIdx.x + stride];
		        //max = sdata[2 * threadIdx.x];
		    }
		}
	}

	if (operation == 3) {
		//min = sdata[0];
		for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
	    	__syncthreads();
		    //if (min > sdata[2 * threadIdx.x + stride]) {
		    if (sdata[2 * threadIdx.x] > sdata[2 * threadIdx.x + stride]) {
		        sdata[2 * threadIdx.x] = sdata[2 * threadIdx.x + stride];
		        //min = sdata[2 * threadIdx.x];
		    }
		}
	}


   if (threadIdx.x == 0) {
            input[blockIdx.x] = sdata[0];
            printf("Result: %d\n", input[0]);
        }
}


int main() {
    // Allocate memory
    int *input;
    // int x;
    hipMallocManaged(&input, SIZE * sizeof(int));

    // Initialize inputs
    for (int i = 0; i < SIZE; i++) {
        input[i] = 1;
    }
    int x = (SIZE + BLOCKSIZE * 2 - 1) / (BLOCKSIZE * 2);

	// Check if SIZE is a multiple of BLOCK_SIZE 
	if (SIZE % BLOCKSIZE != 0) { 
		if (SIZE > BLOCKSIZE) { 
			x += 1; 
			printf("Number of blocks (with extra): %d\n", x); 
		} 
	} else { 
			printf("Number of blocks (perfectly divisible): %d\n", x); 
		}
   
    // Launch the kernel with the calculated number of blocks
    //Reduction<<<x, BLOCKSIZE>>>(input, 0); // sum
   	// Reduction<<<x, BLOCKSIZE>>>(input, 1); // product
    //Reduction<<<x, BLOCKSIZE>>>(input, 2); // max
    //Reduction<<<x, BLOCKSIZE>>>(input, 3); // min

    hipDeviceSynchronize();

    // Sum the results from each block - uncomment when operation is 0
    /*
    int sum = 0;
    for (int i = 0; i < x; i++) {
        sum += input[i];
    }
    */
    
	// Multiply the results from each block - uncomment when operation is 1 
	/*
	int product = 1;
	for (int i = 0; i < x; i++) { 
	 	product *= input[i]; 
	}
	*/

	// Find the max from all blocks - uncomment when operation is 2
	/*
	int max = input[0];
	for (int i=0; i < SIZE; i++){
		if (max < input[i]) {
			max = input[i];
		}
	}
	*/

	// Find the min from all blocks - uncomment when operation is 3
	int min = input[0];
		for (int i=0; i < SIZE; i++){
			if (min > input[i]) {
				min = input[i];
			}
		}

	
	 
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    //printf("Final Sum: %d\n", sum); // uncomment when operation is 0
    //printf("Final Product: %d\n", product); // uncomment when operation is 1
    //printf("Final Max: %d\n", max); // uncomment when operation is 2
    printf("Final Min: %d\n", min); // uncomment when operation is 3
    
    //printf("Final Sum: %d\n", input[0]);

    // Free memory
    hipFree(input);

    return 0;
}
