
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 4096 // array size
#define BLOCKSIZE 1024 // number of threads per block

// Sources: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

// multi-kernel for being able to handle an array larger than 2048
// gpu can't return from the kernel, so reduction can be broken down into blocks

__global__ void Reduction(int* input, int operation){
	__shared__ int sdata[SIZE];
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	// Initialize shared memory with values from global memory
	sdata[threadIdx.x] = input[i]; // reads in the first 1000 elements
	sdata[threadIdx.x + 1024] = input[i+1024];
	/*
	 // Initialize shared memory with values from global memory
    if (i < SIZE) {
        sdata[threadIdx.x] = input[i];
    } else {
        sdata[threadIdx.x] = 0;  // Ensure unused threads have 0 in shared memory
    }
    */
	__syncthreads();
	
// Sum Operation
	if (operation == 0) {
		// int output = 0;
		for (unsigned int stride = 1;  stride <= blockDim.x; stride *= 2) {
		 	
		 __syncthreads();
			
		 	if (threadIdx.x % stride == 0) {
		 	//	input[2*threadIdx.x] += input[2*threadIdx.x+stride];
		 	// sdata[2*threadIdx.x] += sdata[2*threadIdx.x+stride];
		 		sdata[2*threadIdx.x] += sdata[2*threadIdx.x + stride];

		 	}
		 		
		}	

		if (threadIdx.x == 0) {
			input[0] = sdata[0];
			// printf("Sum: %d\n", input[blockDim.x]);

			printf("Sum: %d\n", input[0]);
		}
	// printf("Sum: %d\n", input[blockDim.x]);

	}

}

int main(){
	// allocate memory
	int *input;
	// int x=1; // Number of blocks we're launching
	int x;
	
	hipMallocManaged(&input, SIZE*sizeof(int));

  	// initialize inputs
  	for (int i = 0; i < SIZE; i++) {
  		input[i] = 1;
   	}


	if (SIZE >= 1) {
		x =  (SIZE + BLOCKSIZE)  / BLOCKSIZE; // calculating the new number of blocks for next iteration 
		Reduction<<<x/2, BLOCKSIZE>>>(input, 0);

	}


	/*
	 // Check if SIZE is a multiple of BLOCK_SIZE
    if (SIZE % BLOCKSIZE != 0) {
      // If not a perfect multiple, calculate the number of blocks needed
      if (SIZE > BLOCKSIZE) {
        x = SIZE / BLOCKSIZE + 1; // Add 1 if not perfectly divisible
        printf("Number of blocks (with extra): %d\n", x);
      }
    } else {
      x = SIZE / BLOCKSIZE; // Perfectly divisible case
      printf("Number of blocks (perfectly divisible): %d\n", x);
    }
    */
	
   
  	// Launch the kernel with the calculated number of blocks
    Reduction<<<x/2, BLOCKSIZE>>>(input, 0);
 	
 	hipDeviceSynchronize(); 
 	
 	printf("%s\n", hipGetErrorString(hipGetLastError()));

 	printf("Final Sum: %d\n", input[0]);

	hipFree(input);

	return 0;
	
}
