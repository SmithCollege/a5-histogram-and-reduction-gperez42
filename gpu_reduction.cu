
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 50
#define BLOCKSIZE 1

//  multi-kernel for being able to handle an array larger than 2048
// gpu can't return from the kernel

__global__ void Reduction(int* input, int operation){

// Sum Operation
	if (operation == 0) {
		for (unsigned int stride = 1;  stride <= blockDim.x; stride *= 2) {
		 	
		 __syncthreads();
			
		 	if (threadIdx.x % stride == 0) {
		 		input[2*threadIdx.x] += input[2*threadIdx.x+stride];
		 	}
		 		
		}	

		if (threadIdx.x == 0) {
			printf("Sum: %d\n", input[0]);
		}
	
	}

}

int main(){
	// allocate memory
	// int *input, *output; 
	int *input;
	int x; // Number of blocks we're launching
	
	hipMallocManaged(&input, SIZE*sizeof(int));
	// cudaMallocManaged(&output, sizeof(int) * SIZE);

  	// initialize inputs
  	for (int i = 0; i < SIZE; i++) {
  		input[i] = 1;
   	}

	// Reduction(input, output, 0);
	// printf("Output: %d", Reduction(input, output, 0));

	/*
	if (SIZE < 2048) {
		Reduction<<<1, SIZE>>>(input, 0);
	}
	else{
		BLOCKSIZE = SIZE/2; 
		Reduction<<<BLOCKSIZE, SIZE>>>(input, 0);

	}
	*/

	 // Check if SIZE is a multiple of BLOCK_SIZE
    if (SIZE % BLOCKSIZE != 0) {
      // If not a perfect multiple, calculate the number of blocks needed
      if (SIZE > BLOCKSIZE) {
        x = SIZE / BLOCKSIZE + 1; // Add 1 if not perfectly divisible
        printf("Number of blocks (with extra): %d\n", x);
      }
    } else {
      x = SIZE / BLOCKSIZE; // Perfectly divisible case
      printf("Number of blocks (perfectly divisible): %d\n", x);
    }
    
  	// Launch the kernel with the calculated number of blocks
    Reduction<<<BLOCKSIZE, SIZE>>>(input, 0);
 	
 	hipDeviceSynchronize(); 
 	
 	printf("%s\n", hipGetErrorString(hipGetLastError()));

	hipFree(input);
	// cudaFree(output);

	return 0;
	
}
