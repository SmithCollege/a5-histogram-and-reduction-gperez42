
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 50

//  multi-kernel for being able to handle an array larger than 2048

__global__ void Reduction(int* input, int* output, int operation){
	// Stride is distance to the next value being
	// accumulated into the threads mapped position
	// in the partialSum[] aray
	/*
	for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
		 __syncthreads();
		
		 if (t % stride == 0) {
			 partialSum[2*t]+= partialSum[2*t+stride];
		}
	}
	*/

/*
	// Sum Operation
	if (operation == 0){
		for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
			if (SIZE % stride == 0){
				//output[stride] = input[stride] + input[stride*2];
				//output[2*SIZE] = input[2*SIZE] + input[2*SIZE*stride];
				output[2*SIZE] = input[2*SIZE] + input[2*SIZE*stride];

			}
				
		}
	printf("Sum: %d\n", output[SIZE]);
	// printf("Output: %d", output[stride]);
	}	
	//return output[SIZE];
*/


	if (operation == 0) {
		output[SIZE] = 0;
		for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) {
		 	
		 __syncthreads();

	
			if (SIZE < stride) {
		 		output[SIZE] += input[SIZE+stride];
		 	}
		 
		 /*
		 	if (threadIdx.x < stride) {
		 		output[threadIdx.x] += output[threadIdx.x+stride];
		 	}
		 */		
		}	
	
	printf("Sum: %d\n", output[SIZE]);
	
	}


}

int main(){
	// allocate memory
	int *input, *output; 
	// int sum;	
	hipMallocManaged(&input, sizeof(int) * SIZE);
	hipMallocManaged(&output, sizeof(int));

  	// initialize inputs
  	for (int i = 0; i < SIZE; i++) {
  		input[i] = 1;
   	}
	
	// Reduction(input, output, 0);
	// printf("Output: %d", Reduction(input, output, 0));
	Reduction<<<1, SIZE>>>(input, output, 0);

	/*
	  // check results
 	 for (int i = 0; i < SIZE; i++) {
    	printf("%d ", output[i]);
  	 }
 	 printf("\n");
 	*/ 
 	printf("%s\n", hipGetErrorString(hipGetLastError()));

	hipFree(input);
	hipFree(output);

	return 0;
	
}
