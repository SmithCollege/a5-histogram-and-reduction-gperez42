
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 50

int Reduction(int* input, int* output, int operation){
	// Stride is distance to the next value being
	// accumulated into the threads mapped position
	// in the partialSum[] aray
	/*
	for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
		 __syncthreads();
		
		 if (t % stride == 0) {
			 partialSum[2*t]+= partialSum[2*t+stride];
		}
	}
	*/

	// Sum Operation
	if (operation == 0){
		for (unsigned int stride = 1; stride <= SIZE; stride *= 2) {
			if (SIZE % stride == 0){
				// output[stride] = input[stride] + input[stride*2];
				output[2*SIZE] = input[2*SIZE] + input[2*SIZE*stride];
			}
				
		}
	// printf("Output: %d", output[stride]);
	}	
}

int main(){
	// allocate memory
	int* input = (int*) malloc(sizeof(int) * SIZE);
	int* output = (int*) malloc(sizeof(int) * SIZE);

  	// initialize inputs
  	for (int i = 0; i < SIZE; i++) {
  		input[i] = 1;
   	}
	
	//Reduction(input, output, 0);
	printf("Output: %d", Reduction(input, output, 0));

	/*
	  // check results
 	 for (int i = 0; i < SIZE; i++) {
    	printf("%d ", output[i]);
  	 }
 	 printf("\n");
 	*/ 
	

	return 0;
	
}
