
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 4096 // array size
#define BLOCKSIZE 1024 // number of threads per block

// Sources: https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

// multi-kernel for being able to handle an array larger than 2048
// gpu can't return from the kernel, so reduction can be broken down into blocks

__global__ void Reduction(int* input, int operation){
	__shared__ int sdata[SIZE];
	
	// unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	// each thread handles two elements per iteration when initializing shared memory
     unsigned int i = blockIdx.x* blockDim.x + threadIdx.x;

	
	// Initialize shared memory with values from global memory
	sdata[threadIdx.x] = input[i]; // reads in the first 1024 elements
	// sdata[threadIdx.x + 1024] = input[i+1024];

	// sdata[threadIdx.x] = input[i] + ((i + blockDim.x) < SIZE ? input[i + blockDim.x] : 0);
    sdata[threadIdx.x + BLOCKSIZE] = input[i+BLOCKSIZE]; // reads in the next 1024 elements

/*
		if (i < SIZE) { 
			sdata[threadIdx.x] = input[i] + ((i + blockDim.x) < SIZE ? input[i + blockDim.x] : 0); 
		} 
		else { 
			sdata[threadIdx.x] = 0; 
		}
*/
	
	/*
	 // Initialize shared memory with values from global memory
    if (i < SIZE) {
        sdata[threadIdx.x] = input[i];
    } else {
        sdata[threadIdx.x] = 0;  // Ensure unused threads have 0 in shared memory
    }
    */
	__syncthreads();
	
// Sum Operation
	if (operation == 0) {
		// int output = 0;
		for (unsigned int stride = 1;  stride <= blockDim.x; stride *= 2) {
		 	
		 __syncthreads();
			
		 	if (threadIdx.x % stride == 0) {
		 	//	input[2*threadIdx.x] += input[2*threadIdx.x+stride];
		 	// sdata[2*threadIdx.x] += sdata[2*threadIdx.x+stride];
		 		sdata[2*threadIdx.x] += sdata[2*threadIdx.x + stride];

		 	}
		 		
		}	

		if (threadIdx.x == 0) {
			input[0] = sdata[0];
			// printf("Sum: %d\n", input[blockDim.x]);

			printf("Sum: %d\n", input[0]);
		}
	// printf("Sum: %d\n", input[blockDim.x]);

	}

}

int main(){
	// allocate memory
	int *input;
    //int x=1; // Number of blocks we're launching
	int x=2;
	
	hipMallocManaged(&input, SIZE*sizeof(int));

  	// initialize inputs
  	for (int i = 0; i < SIZE; i++) {
  		input[i] = 1;
   	}



	if (SIZE >= 1) {
		//x =  (SIZE + BLOCKSIZE) / BLOCKSIZE; // calculating the new number of blocks for next iteration 
		//Reduction<<<x/2, BLOCKSIZE>>>(input, 0);

		x = (SIZE + BLOCKSIZE * 2 - 1) / (BLOCKSIZE * 2);
		
		if (x > 1) {
			Reduction<<<x/2, BLOCKSIZE>>>(input, 0);
		}
		else {
			Reduction<<<x, BLOCKSIZE>>>(input, 0);

		}


	}


	/*
	 // Check if SIZE is a multiple of BLOCK_SIZE
    if (SIZE % BLOCKSIZE != 0) {
      // If not a perfect multiple, calculate the number of blocks needed
      if (SIZE > BLOCKSIZE) {
        x = SIZE / BLOCKSIZE + 1; // Add 1 if not perfectly divisible
        printf("Number of blocks (with extra): %d\n", x);
      }
    } else {
      x = SIZE / BLOCKSIZE; // Perfectly divisible case
      printf("Number of blocks (perfectly divisible): %d\n", x);
    }
    */
	
   
  	// Launch the kernel with the calculated number of blocks
    Reduction<<<x/2, BLOCKSIZE>>>(input, 0);
 	
    hipDeviceSynchronize(); 

	/*
 	// Sum the results from each block 
 	int sum = 0; 
 	for (int i = 0; i < x; i++) { 
 		printf("before %d\n", sum);
 		sum += input[i]; 
 		printf("after %d\n", sum);

 	}
 	*/
 	
 	printf("%s\n", hipGetErrorString(hipGetLastError()));

 	printf("Final Sum: %d\n", input[0]);
 	// printf("Final Sum: %d\n", sum);


	hipFree(input);

	return 0;
	
}
